#include "hip/hip_runtime.h"
#include "process_input_frame.cuh"
#include <iostream>
#include <opencv2/imgproc.hpp>

// Resize UV
//

__global__ void resizeUV(void* uInput, void* vInput, float* uOutput, float* vOutput, int inputWidth, int inputHeight, int outputWidth, int outputHeight, int type)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outputWidth && y < outputHeight)
    {
        float scaleX = (float)(inputWidth) / outputWidth;
        float scaleY = (float)(inputHeight) / outputHeight;

        float srcX = x * scaleX;
        float srcY = y * scaleY;

        int x1 = (int)srcX;
        int y1 = (int)srcY;
        int x2 = fminf(x1 + 1, inputWidth - 1);
        int y2 = fminf(y1 + 1, inputHeight - 1);

        float dx = srcX - x1;
        float dy = srcY - y1;

        float val1, val2;

        if (type == CV_32F)
        {
            val1 = ((float*)uInput)[y1 * inputWidth + x1] * (1 - dx) + ((float*)uInput)[y1 * inputWidth + x2] * dx;
            val2 = ((float*)uInput)[y2 * inputWidth + x1] * (1 - dx) + ((float*)uInput)[y2 * inputWidth + x2] * dx;
        }
        else if (type == CV_16U)
        {
            val1 = ((unsigned short*)uInput)[y1 * inputWidth + x1] * (1 - dx) + ((unsigned short*)uInput)[y1 * inputWidth + x2] * dx;
            val2 = ((unsigned short*)uInput)[y2 * inputWidth + x1] * (1 - dx) + ((unsigned short*)uInput)[y2 * inputWidth + x2] * dx;
        }
        else if (type == CV_8U)
        {
            val1 = ((unsigned char*)uInput)[y1 * inputWidth + x1] * (1 - dx) + ((unsigned char*)uInput)[y1 * inputWidth + x2] * dx;
            val2 = ((unsigned char*)uInput)[y2 * inputWidth + x1] * (1 - dx) + ((unsigned char*)uInput)[y2 * inputWidth + x2] * dx;
        }

        uOutput[y * outputWidth + x] = val1 * (1 - dy) + val2 * dy;

        if (type == CV_32F)
        {
            val1 = ((float*)vInput)[y1 * inputWidth + x1] * (1 - dx) + ((float*)vInput)[y1 * inputWidth + x2] * dx;
            val2 = ((float*)vInput)[y2 * inputWidth + x1] * (1 - dx) + ((float*)vInput)[y2 * inputWidth + x2] * dx;
        }
        else if (type == CV_16U)
        {
            val1 = ((unsigned short*)vInput)[y1 * inputWidth + x1] * (1 - dx) + ((unsigned short*)vInput)[y1 * inputWidth + x2] * dx;
            val2 = ((unsigned short*)vInput)[y2 * inputWidth + x1] * (1 - dx) + ((unsigned short*)vInput)[y2 * inputWidth + x2] * dx;
        }
        else if (type == CV_8U)
        {
            val1 = ((unsigned char*)vInput)[y1 * inputWidth + x1] * (1 - dx) + ((unsigned char*)vInput)[y1 * inputWidth + x2] * dx;
            val2 = ((unsigned char*)vInput)[y2 * inputWidth + x1] * (1 - dx) + ((unsigned char*)vInput)[y2 * inputWidth + x2] * dx;
        }

        vOutput[y * outputWidth + x] = val1 * (1 - dy) + val2 * dy;
    }
}

void resizeImage(cv::Mat inputs[], float*& devUOutput, float*& devVOutput, cv::Size size, int type)
{
    void* devUInput;
    void* devVInput;

    size_t inputSize = inputs[0].total() * inputs[0].elemSize();
    size_t outputSize = size.area() * sizeof(float);

    hipMalloc(&devUInput, inputSize);
    hipMalloc(&devVInput, inputSize);
    hipMalloc(&devUOutput, outputSize);
    hipMalloc(&devVOutput, outputSize);

    hipMemcpy(devUInput, inputs[0].data, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(devVInput, inputs[1].data, inputSize, hipMemcpyHostToDevice);

    int blockWidth = 16;
    int blockHeight = 8;
    dim3 gridDim((size.width - 1 + blockWidth) / blockWidth, (size.height - 1 + blockHeight) / blockHeight);
    dim3 blockDim(blockWidth, blockHeight);

    if (type == CV_32F || type == CV_16U || type == CV_8U)
    {
        resizeUV<<<gridDim, blockDim>>>(devUInput, devVInput, devUOutput, devVOutput, inputs[0].cols, inputs[0].rows, size.width, size.height, type);
    }
    else
    {
        hipFree(devUInput);
        hipFree(devVInput);
        hipFree(devUOutput);
        hipFree(devVOutput);
        inputs[0].release();
        inputs[1].release();

        throw std::invalid_argument("Unsupported CV type");
    }

    hipDeviceSynchronize();

    hipFree(devUInput);
    hipFree(devVInput);
    inputs[0].release();
    inputs[1].release();

    hipError_t state = hipGetLastError();
    if (state != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(state));
    }
}

// Resize Y
//

__global__ void resizeY(void* yInput, float* yOutput, int inputWidth, int inputHeight, int outputWidth, int outputHeight, int type)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outputWidth && y < outputHeight)
    {
        float scaleX = (float)(inputWidth) / outputWidth;
        float scaleY = (float)(inputHeight) / outputHeight;

        float srcX = x * scaleX;
        float srcY = y * scaleY;

        int x1 = (int)srcX;
        int y1 = (int)srcY;
        int x2 = fminf(x1 + 1, inputWidth - 1);
        int y2 = fminf(y1 + 1, inputHeight - 1);

        float dx = srcX - x1;
        float dy = srcY - y1;

        float val1, val2;
        if (type == CV_32F)
        {
            val1 = ((float*)yInput)[y1 * inputWidth + x1] * (1 - dx) + ((float*)yInput)[y1 * inputWidth + x2] * dx;
            val2 = ((float*)yInput)[y2 * inputWidth + x1] * (1 - dx) + ((float*)yInput)[y2 * inputWidth + x2] * dx;
        }
        else if (type == CV_16U)
        {
            val1 = ((unsigned short*)yInput)[y1 * inputWidth + x1] * (1 - dx) + ((unsigned short*)yInput)[y1 * inputWidth + x2] * dx;
            val2 = ((unsigned short*)yInput)[y2 * inputWidth + x1] * (1 - dx) + ((unsigned short*)yInput)[y2 * inputWidth + x2] * dx;
        }
        else if (type == CV_8U)
        {
            val1 = ((unsigned char*)yInput)[y1 * inputWidth + x1] * (1 - dx) + ((unsigned char*)yInput)[y1 * inputWidth + x2] * dx;
            val2 = ((unsigned char*)yInput)[y2 * inputWidth + x1] * (1 - dx) + ((unsigned char*)yInput)[y2 * inputWidth + x2] * dx;
        }

        yOutput[y * outputWidth + x] = val1 * (1 - dy) + val2 * dy;
    }
}


// Merge
//


__global__ void mergeYUV(const void* yChannel, const float* uChannel, const float* vChannel, float3* yuvChannels, int width, int height, float colorScale, int type)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int idx = y * width + x;
        float3 pixel;

        if (type == CV_32F)
        {
            pixel.x = ((float*)yChannel)[idx] * colorScale;
        }
        else if (type == CV_16U)
        {
            pixel.x = ((unsigned short*)yChannel)[idx] * colorScale;
        }
        else if (type == CV_8U)
        {
            pixel.x = ((unsigned char*)yChannel)[idx] * colorScale;
        }
        pixel.y = uChannel[idx] * colorScale;
        pixel.z = vChannel[idx] * colorScale;

        yuvChannels[idx] = pixel;
    }
}

void mergeFrame(cv::Mat& yInput, float* devU, float* devV, float3*& devYUV, cv::Size outputSize, int yType, float colorScale)
{
    int width = outputSize.width;
    int height = outputSize.height;

    void* devY;

    size_t yInputSize = yInput.total() * yInput.elemSize();
    size_t uvInputSize = outputSize.area() * sizeof(float);
    size_t yuvOutputSize = outputSize.area() * sizeof(float3);

    hipMalloc(&devY, yInputSize);
    hipMalloc(&devYUV, yuvOutputSize);

    hipMemcpy(devY, yInput.data, yInputSize, hipMemcpyHostToDevice);

    int blockWidth = 16;
    int blockHeight = 8;
    dim3 gridDim((width - 1 + blockWidth) / blockWidth, (height - 1 + blockHeight) / blockHeight);
    dim3 blockDim(blockWidth, blockHeight);

    if (yInput.size() != outputSize)
    {
        float* devRescaledY;
        hipMalloc(&devRescaledY, outputSize.area() * sizeof(float));

        if (yType == CV_32F || yType == CV_16U || yType == CV_8U)
        {
            resizeY<<<gridDim, blockDim>>>(devY, devRescaledY, yInput.cols, yInput.rows, outputSize.width, outputSize.height, yType);
        }
        else
        {
            hipFree(devY);
            hipFree(devU);
            hipFree(devV);
            hipFree(devRescaledY);
            hipFree(devYUV);
            throw std::invalid_argument("Unsupported CV type");
        }

        hipDeviceSynchronize();

        mergeYUV<<<gridDim, blockDim>>>(devRescaledY, devU, devV, devYUV, width, height, colorScale, CV_32F);

        hipDeviceSynchronize();

        hipFree(devRescaledY);
    }
    else
    {
        if (yType == CV_32F || yType == CV_16U || yType == CV_8U)
        {
            mergeYUV<<<gridDim, blockDim>>>(devY, devU, devV, devYUV, width, height, colorScale, yType);
        }
        else
        {
            hipFree(devY);
            hipFree(devU);
            hipFree(devV);
            hipFree(devYUV);
            yInput.release();
            throw std::invalid_argument("Unsupported CV type");
        }

        hipDeviceSynchronize();
    }

    hipFree(devY);
    hipFree(devU);
    hipFree(devV);
    yInput.release();

    hipError_t state = hipGetLastError();
    if (state != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(state));
    }
}

// Format depth
//

__global__ void normalize(void* src, float* dst, float scale, float near, float far, int rows, int cols, int type, bool hasInvalidDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows)
    {
        int idx = y * cols + x;
        float depth = 0.0f;

        switch (type)
        {
            case CV_32F:
                depth = ((float*)src)[idx] * scale;
                break;
            case CV_16U:
                depth = ((unsigned short*)src)[idx] * scale;
                break;
            case CV_8U:
                depth = ((unsigned char*)src)[idx] * scale;
                break;
        }

        if (far >= 1000.f)
        {
            depth = near / depth;
        }
        else
        {
            depth = far * near / (near + depth * (far - near));
        }

        if (hasInvalidDepth && depth == 0)
        {
            depth = NAN;
        }

        dst[idx] = depth;
    }
}

void formatDepth(cv::Mat& src, float*& devDst, cv::Size size, float scale, float near, float far, bool hasInvalidDepth, int type)
{
    int rows = size.height;
    int cols = size.width;

    size_t inputSize = src.total() * src.elemSize();
    size_t outputSize = size.area() * sizeof(float);

    void* devSrc;

    hipMalloc(&devSrc, inputSize);
    hipMalloc(&devDst, outputSize);

    hipMemcpy(devSrc, src.data, inputSize, hipMemcpyHostToDevice);

    int blockWidth = 16;
    int blockHeight = 8;
    dim3 gridDim((cols - 1 + blockWidth) / blockWidth, (rows - 1 + blockHeight) / blockHeight);
    dim3 blockDim(blockWidth, blockHeight);

    if (src.size() != size)
    {
        float* devSrcRescaled;

        hipMalloc(&devSrcRescaled, outputSize);

        if (type == CV_32F || type == CV_16U || type == CV_8U)
        {
            resizeY<<<gridDim, blockDim>>>(devSrc, devSrcRescaled, src.cols, src.rows, cols, rows, type);
        }
        else
        {
            hipFree(devSrc);
            hipFree(devDst);
            hipFree(devSrcRescaled);
            throw std::invalid_argument("Unsupported CV type");
        }

        hipDeviceSynchronize();

        normalize<<<gridDim, blockDim>>>(devSrcRescaled, devDst, scale, near, far, rows, cols, CV_32F, hasInvalidDepth);
    }
    else
    {
        if (type == CV_32F || type == CV_16U || type == CV_8U)
        {
            normalize<<<gridDim, blockDim>>>(devSrc, devDst, scale, near, far, rows, cols, type, hasInvalidDepth);
        }
        else
        {
            hipFree(devSrc);
            hipFree(devDst);
            throw std::invalid_argument("Unsupported CV type");
        }
    }

    hipDeviceSynchronize();

    hipFree(devSrc);

    hipError_t state = hipGetLastError();
    if (state != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(state));
    }
}

// Process output frame
//

__global__ void separateChannelsKernel(float3* src, float* Y, float* U, float* V, int rows, int cols)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * cols + x;

    if (y < rows && x < cols)
    {
        float3 pixel = src[idx];

        Y[idx] = pixel.x;
        U[idx] = pixel.y;
        V[idx] = pixel.z;
    }
}

__global__ void quantizationKernel(float* src, void* dst, int rows, int cols, int cv_depth, unsigned max_val)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * cols + x;

    if (y < rows && x < cols)
    {
        float pixel = src[idx];

        switch (cv_depth)
        {
            case CV_8U:
                ((unsigned char*)dst)[idx] = pixel * max_val;
                break;
            case CV_16U:
                ((unsigned short*)dst)[idx] = pixel * max_val;
                break;
            case CV_32F:
                ((float*)dst)[idx] = pixel * max_val;
                break;
        }
    }
}

void quantization(float3* devSrc, cv::Mat& dstY, cv::Mat& dstU, cv::Mat& dstV, int rows, int cols, cv::Size outputSize, int cv_depth, unsigned max_val)
{
    size_t dstSizeY, dstSizeUV;
    cv::Size uvSize(outputSize.width / 2, outputSize.height / 2);

    switch (cv_depth)
    {
        case CV_8U:
            dstSizeY = outputSize.height * outputSize.width * sizeof(unsigned char);
            dstSizeUV = uvSize.height * uvSize.width * sizeof(unsigned char);
            break;
        case CV_16U:
            dstSizeY = outputSize.height * outputSize.width * sizeof(unsigned short);
            dstSizeUV = uvSize.height * uvSize.width * sizeof(unsigned short);
            break;
        case CV_32F:
            dstSizeY = outputSize.height * outputSize.width * sizeof(float);
            dstSizeUV = uvSize.height * uvSize.width * sizeof(float);
            break;
        default:
            throw std::invalid_argument("unsupported CV depth");
    }

    float *devY, *devU, *devV;
    hipMalloc(&devY, rows * cols * sizeof(float));
    hipMalloc(&devU, rows * cols * sizeof(float));
    hipMalloc(&devV, rows * cols * sizeof(float));

    int blockWidth = 16;
    int blockHeight = 8;

    dim3 blockDim(blockWidth, blockHeight);

    dim3 gridDim((cols + blockWidth - 1) / blockWidth, (rows + blockHeight - 1) / blockHeight);
    dim3 resizeGridDimUV((uvSize.width + blockWidth - 1) / blockWidth, (uvSize.height + blockHeight - 1) / blockHeight);

    separateChannelsKernel<<<gridDim, blockDim>>>(devSrc, devY, devU, devV, rows, cols);

    hipDeviceSynchronize();

    void* devDstY;
    void* devDstU;
    void* devDstV;
    hipMalloc(&devDstY, dstSizeY);
    hipMalloc(&devDstU, dstSizeUV);
    hipMalloc(&devDstV, dstSizeUV);

    float* resizedU, * resizedV;

    hipMalloc(&resizedU, uvSize.height * uvSize.width * sizeof(float));
    hipMalloc(&resizedV, uvSize.height * uvSize.width * sizeof(float));

    resizeUV<<<resizeGridDimUV, blockDim>>>(devU, devV, resizedU, resizedV, cols, rows, uvSize.width, uvSize.height, CV_32F);

    if (cv::Size(cols, rows) != outputSize)
    {
        float *resizedY;
        hipMalloc(&resizedY, outputSize.height * outputSize.width * sizeof(float));

        dim3 resizeGridDimY((outputSize.width + blockWidth - 1) / blockWidth, (outputSize.height + blockHeight - 1) / blockHeight);

        resizeY<<<resizeGridDimY, blockDim>>>(devY, resizedY, cols, rows, outputSize.width, outputSize.height, CV_32F);

        hipDeviceSynchronize();

        quantizationKernel<<<resizeGridDimY, blockDim>>>(resizedY, devDstY, outputSize.height, outputSize.width, cv_depth, max_val);

        hipFree(resizedY);
    }
    else
    {
        quantizationKernel<<<gridDim, blockDim>>>(devY, devDstY, outputSize.height, outputSize.width, cv_depth, max_val);
    }

    quantizationKernel<<<resizeGridDimUV, blockDim>>>(resizedU, devDstU, uvSize.height, uvSize.width, cv_depth, max_val);
    quantizationKernel<<<resizeGridDimUV, blockDim>>>(resizedV, devDstV, uvSize.height, uvSize.width, cv_depth, max_val);

    hipFree(resizedU);
    hipFree(resizedV);

    hipDeviceSynchronize();

    switch (cv_depth)
    {
        case CV_8U:
            hipMemcpy(dstY.ptr<unsigned char>(), devDstY, dstSizeY, hipMemcpyDeviceToHost);
            hipMemcpy(dstU.ptr<unsigned char>(), devDstU, dstSizeUV, hipMemcpyDeviceToHost);
            hipMemcpy(dstV.ptr<unsigned char>(), devDstV, dstSizeUV, hipMemcpyDeviceToHost);
            break;
        case CV_16U:
            hipMemcpy(dstY.ptr<unsigned short>(), devDstY, dstSizeY, hipMemcpyDeviceToHost);
            hipMemcpy(dstU.ptr<unsigned short>(), devDstU, dstSizeUV, hipMemcpyDeviceToHost);
            hipMemcpy(dstV.ptr<unsigned short>(), devDstV, dstSizeUV, hipMemcpyDeviceToHost);
            break;
        case CV_32F:
            hipMemcpy(dstY.ptr<float>(), devDstY, dstSizeY, hipMemcpyDeviceToHost);
            hipMemcpy(dstU.ptr<float>(), devDstU, dstSizeUV, hipMemcpyDeviceToHost);
            hipMemcpy(dstV.ptr<float>(), devDstV, dstSizeUV, hipMemcpyDeviceToHost);
            break;
    }

    hipFree(devY);
    hipFree(devU);
    hipFree(devV);
    hipFree(devDstY);
    hipFree(devDstU);
    hipFree(devDstV);

    hipError_t state = hipGetLastError();
    bool error = state != hipSuccess;
    if (error)
        throw std::runtime_error(hipGetErrorString(state));
}