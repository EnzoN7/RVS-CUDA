#include "hip/hip_runtime.h"
#include "blend_img.cuh"

struct ConstantParams
{
    float blendingExponent;
    int imgHeight;
    int imgWidth;
    int numImages;
    float3 emptyColor;
};

__constant__ ConstantParams constParams;

__device__ float3 operator*(float a, float3 b)
{
    return make_float3(a * b.x, a * b.y, a * b.z);
}

__device__ float3 operator+(float3 a, float3 b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator/(float3 a, float b)
{
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__global__ void blendImagesKernel
(
    float3* blendedColor, float* blendedValidity, float* blendedDepth,
    float3* devColor, float* devValidity, float* devDepth,
    float3* outputColor, float* outputValidity, float* outputDepth
)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx >= constParams.imgWidth || ty >= constParams.imgHeight)
        return;

    int threadId = ty * constParams.imgWidth + tx;

    float sumWeights = 0.0f;
    float inpaintedDepthSumWeights = 0.0f;
    float3 inpaintedDepthColor = make_float3(0, 0, 0);
    float3 finalColor = make_float3(0, 0, 0);

    float quality[2] =
    {
        blendedValidity[threadId] / blendedDepth[threadId],
        devValidity[threadId] / devDepth[threadId]
    };
    bool isValidDepth[2] =
    {
            quality[0] > 0 && !(blendedDepth[threadId] > 0.0f),
            quality[1] > 0 && !(devDepth[threadId] > 0.0f)
    };
    float3 color[2] =
    {
        blendedColor[threadId],
        devColor[threadId]
    };

    for (int i = 0; i < constParams.numImages; i++)
    {
        if (quality[i] > 0)
        {
            if (constParams.blendingExponent > 1.0f)
                quality[i] = powf(quality[i], constParams.blendingExponent);

            if (isValidDepth[i])
            {
                sumWeights += quality[i];
                finalColor = finalColor + (quality[i] * color[i]);
            }
            else
            {
                inpaintedDepthSumWeights += quality[i];
                inpaintedDepthColor = inpaintedDepthColor + (quality[i] * color[i]);
            }
        }
    }

    outputDepth[threadId] = (quality[0] > quality[1]) ? blendedDepth[threadId] : devDepth[threadId];
    outputValidity[threadId] = (blendedValidity[threadId] > devValidity[threadId]) ? blendedValidity[threadId] : devValidity[threadId];

    if (sumWeights == 0)
    {
        if (inpaintedDepthSumWeights == 0.0f)
        {
            outputColor[threadId] = constParams.emptyColor;
        }
        else
        {
            outputColor[threadId] = inpaintedDepthColor / inpaintedDepthSumWeights;
        }
    }
    else
    {
        outputColor[threadId] = finalColor / sumWeights;
    }
}

void blendImages
(
    float3* devBlendedColor, float* devBlendedValidity, float* devBlendedDepth,
    float3* devColor, float* devValidity, float* devDepth, cv::Size outputSize,
    cv::Vec3f emptyColor, float3*& devOutputColor, float*& devOutputValidity, float*& devOutputDepth, float blendingExponent
)
{
    int imgHeight = outputSize.height;
    int imgWidth = outputSize.width;
    int imgSize = imgWidth * imgHeight;
    int numImages = 2;

    float3 emptyColor3 = make_float3(emptyColor[0], emptyColor[1], emptyColor[2]);

    ConstantParams params = { blendingExponent, imgHeight, imgWidth, numImages, emptyColor3 };
    hipMemcpyToSymbol(HIP_SYMBOL(constParams), &params, sizeof(ConstantParams));

    hipMalloc(&devOutputColor, imgSize * sizeof(float3));
    hipMalloc(&devOutputValidity, imgSize * sizeof(float));
    hipMalloc(&devOutputDepth, imgSize * sizeof(float));

    int blockWidth = 8;
    int blockHeight = 16;
    dim3 gridDim((imgWidth - 1 + blockWidth) / blockWidth, (imgHeight - 1 + blockHeight) / blockHeight);
    dim3 blockDim(blockWidth, blockHeight);

    blendImagesKernel <<<gridDim, blockDim>>> (
        devBlendedColor, devBlendedValidity, devBlendedDepth,
        devColor, devValidity, devDepth,
        devOutputColor, devOutputValidity, devOutputDepth);

    hipDeviceSynchronize();

    hipFree(devBlendedColor);
    hipFree(devBlendedValidity);
    hipFree(devBlendedDepth);

    hipFree(devColor);
    hipFree(devValidity);
    hipFree(devDepth);

    hipError_t state = hipGetLastError();
    bool error = state != hipSuccess;
    if (error)
        throw std::runtime_error(hipGetErrorString(state));
}
