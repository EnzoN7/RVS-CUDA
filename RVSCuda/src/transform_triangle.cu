#include "hip/hip_runtime.h"
#include "transform_triangle.cuh"
#include <iostream>

/**
* Auteur : Enzo DI MARIA
*/

/**
 * @brief A constant structure to hold the dimensions of the image (or the grid) being processed.
 */
__constant__ int imgDim[2];

/**
* @brief  A constant structure to hold the desired size of the output color and depth images.
*/
__constant__ int outputImgDim[2];

/**
 * @brief Computes the squared Euclidean distance between two points in 2D space.
 *
 * @param p1 First point as a float2 containing the x and y coordinates.
 * @param p2 Second point as a float2 containing the x and y coordinates.
 *
 * @return The squared distance between p1 and p2.
 */
__device__ inline float dot(float2 p1, float2 p2)
{
    float dx = p1.x - p2.x;
    float dy = p1.y - p2.y;
    return dx * dx + dy * dy;
}

/**
 * @brief Evaluates the validity of a triangle based on the lengths of its sides.
 *
 * @param posA The position of the first vertex of the triangle.
 * @param posB The position of the second vertex of the triangle.
 * @param posC The position of the third vertex of the triangle.
 *
 * @return A quality score for the triangle, ranging from 1 to 10000.
 */
__device__ inline float isTriValid(float2 posA, float2 posB, float2 posC)
{
    float ab = dot(posA, posB);
    float ac = dot(posA, posC);
    float bc = dot(posB, posC);

    float quality = 10000.f - 1000.f * sqrtf(fmaxf(bc, fmaxf(ab, ac)));

    return fminf(10000.f, fmaxf(1.f, quality));
}

struct DepthAndValidity
{
    float depth;
    float validity;
};

union DepthAndValidityUnion
{
    DepthAndValidity dv;
    unsigned long long int encoded;
};

__device__ unsigned long long int encodeDepthAndValidity(float depth, float validity)
{
    DepthAndValidityUnion u;
    u.dv.depth = depth;
    u.dv.validity = validity;
    return u.encoded;
}

__device__ DepthAndValidity decodeDepthAndValidity(unsigned long long int encoded)
{
    DepthAndValidityUnion u;
    u.encoded = encoded;
    return u.dv;
}

/**
 * @brief Processes a single pixel within a triangle, updating color and depth based on barycentric coordinates.
 *
 * @param px The x-coordinate of the pixel to process.
 * @param py The y-coordinate of the pixel to process.
 * @param invArea The precomputed inverse area of the triangle.
 * @param outputColor The global result buffer where the new color will be stored.
 * @param outputDepth The global buffer where the new depth will be stored.
 * @param quality The global buffer indicating the shape quality of the triangles.
 * @param validity The global buffer indicating the shape quality of the triangles.
 * @param triangleValidity The threshold for the validity of triangle shape updates.
 * @param posA The position of the first vertex of the triangle.
 * @param posB The position of the second vertex of the triangle.
 * @param posC The position of the third vertex of the triangle.
 * @param colA The color of the first vertex of the triangle.
 * @param colB The color of the second vertex of the triangle.
 * @param colC The color of the third vertex of the triangle.
 * @param depA The depth of the first vertex of the triangle.
 * @param depB The depth of the second vertex of the triangle.
 * @param depC The depth of the third vertex of the triangle.
 */
__device__ void processPixel
(
    int px, int py, float invArea, float3* outputColor, unsigned long long int* outputDepthValidity, float triangleValidity,
    float2 posA, float2 posB, float2 posC, float3 colA, float3 colB, float3 colC, float depA, float depB, float depC
)
{
    int pixelId = py * outputImgDim[0] + px;

    float offsetPx = (float)px + 0.5f;
    float offsetPy = (float)py + 0.5f;

    float lambda1 = invArea * ((posB.y - posC.y) * (offsetPx - posC.x) + (posC.x - posB.x) * (offsetPy - posC.y));
    if (lambda1 < 0) return;

    float lambda2 = invArea * ((posC.y - posA.y) * (offsetPx - posC.x) + (posA.x - posC.x) * (offsetPy - posC.y));
    if (lambda2 < 0) return;

    float lambda3 = 1.0f - lambda1 - lambda2;
    if (lambda3 < 0) return;

    float depth = fmaf(depA, lambda1, fmaf(depB, lambda2, depC * lambda3));

    unsigned long long int newEncoded = encodeDepthAndValidity(depth, triangleValidity);
    unsigned long long int oldEncoded = atomicExch(&outputDepthValidity[pixelId], newEncoded);

    DepthAndValidity oldDV = decodeDepthAndValidity(oldEncoded);

    float ratio = depth / oldDV.depth;

    if (oldDV.validity * ratio * ratio * ratio > triangleValidity)
    {
        atomicExch(&outputDepthValidity[pixelId], oldEncoded);
        return;
    }

    float3 newColor = make_float3
    (
        fmaf(colA.x, lambda1, fmaf(colB.x, lambda2, colC.x * lambda3)),
        fmaf(colA.y, lambda1, fmaf(colB.y, lambda2, colC.y * lambda3)),
        fmaf(colA.z, lambda1, fmaf(colB.z, lambda2, colC.z * lambda3))
    );

    oldEncoded = atomicExch(&outputDepthValidity[pixelId], newEncoded);
    oldDV = decodeDepthAndValidity(oldEncoded);
    ratio = depth / oldDV.depth;

    if (oldDV.validity * ratio * ratio * ratio > triangleValidity)
    {
        atomicExch(&outputDepthValidity[pixelId], oldEncoded);
        return;
    }

    outputColor[pixelId] = newColor;
}

/**
 * @brief Processes a triangle to update colors and depths of its pixels.
 *
 * @param posA The position of the first vertex of the triangle.
 * @param posB The position of the second vertex of the triangle.
 * @param posC The position of the third vertex of the triangle.
 * @param colA The color of the first vertex of the triangle.
 * @param colB The color of the second vertex of the triangle.
 * @param colC The color of the third vertex of the triangle.
 * @param depA The depth of the first vertex of the triangle.
 * @param depB The depth of the second vertex of the triangle.
 * @param depC The depth of the third vertex of the triangle.
 * @param outputColor The global result buffer where the new color will be stored.
 * @param outputDepthValidity The global buffer where the new depth and validity will be stored.
 */
__device__ void processTriangle
(
    float2 posA, float2 posB, float2 posC, float3 colA, float3 colB, float3 colC, float depA, float depB, float depC, float3* outputColor, unsigned long long int* outputDepthValidity
)
{
    if (depA <= 0 || posA.x <= 0 || isnan(depA) || isnan(posA.x))
        return;

    int xMin = fmaxf(0, floorf(fminf(fminf(posA.x, posB.x), posC.x)));
    int xMax = fminf(outputImgDim[0] - 1, ceilf(fmaxf(fmaxf(posA.x, posB.x), posC.x)));
    int yMin = fmaxf(0, floorf(fminf(fminf(posA.y, posB.y), posC.y)));
    int yMax = fminf(outputImgDim[1] - 1, ceilf(fmaxf(fmaxf(posA.y, posB.y), posC.y)));

    if (yMin >= yMax || xMin >= xMax)
        return;

    float invArea = (posB.y - posC.y) * (posA.x - posC.x) + (posC.x - posB.x) * (posA.y - posC.y);

    if (invArea <= 0.0f)
        return;

    invArea = 1.0f / invArea;

    float triangleValidity = isTriValid(posA, posB, posC);

    for (int dy = yMin; dy <= yMax; ++dy)
        for (int dx = xMin; dx <= xMax; ++dx)
            processPixel(dx, dy, invArea, outputColor, outputDepthValidity, triangleValidity, posA, posB, posC, colA, colB, colC, depA, depB, depC);
}

/**
 * @brief CUDA kernel that colorizes triangles based on depth and position data.
 *
 * The kernel assumes that each thread corresponds to a single pixel, indexed by the 2D block and grid
 * dimensions. It uses shared constants 'imgDim' to represent image dimensions and 'isErp' to indicate if
 * equirectangular projection wrapping is required.
 *
 * @param inputColor The input image containing colors in YUV.
 * @param inputDepth The input image containing depth values.
 * @param inputPositions The positions of triangle vertices.
 * @param outputColor The global result buffer where the new color will be stored.
 * @param outputDepthValidity The global buffer where the new depth and validity will be stored.
 * @param horizontalWarping A boolean constant indicating if the image has a horizontal warping.
 */
__global__ void colorizeTriangleKernel(float3* inputColor, float* inputDepth, float2* inputPositions, float3* outputColor, unsigned long long int* outputDepthValidity, bool horizontalWarping)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int threadId = ty * imgDim[0] + tx;

    if (tx >= imgDim[0] - 1 || ty >= imgDim[1] - 1)
        return;

    int3 triPixelIds = make_int3(threadId + 1 + imgDim[0], threadId + 1, threadId + imgDim[0]);

    float depB = inputDepth[triPixelIds.y];
    if (isnan(depB) || depB <= 0.f)
        return;
    float depC = inputDepth[triPixelIds.z];
    if (isnan(depC) || depC <= 0.f)
        return;

    float2 posB = inputPositions[triPixelIds.y];
    if (isnan(posB.x) || posB.x <= 0.f)
        return;
    float2 posC = inputPositions[triPixelIds.z];
    if (isnan(posC.x) || posC.x <= 0.f)
        return;

    float3 colB = inputColor[triPixelIds.y];
    float3 colC = inputColor[triPixelIds.z];

    processTriangle(inputPositions[threadId], posB, posC, inputColor[threadId], colB, colC, inputDepth[threadId], depB, depC, outputColor, outputDepthValidity);
    processTriangle(inputPositions[triPixelIds.x], posC, posB, inputColor[triPixelIds.x], colC, colB, inputDepth[triPixelIds.x], depC, depB, outputColor, outputDepthValidity);

    if (horizontalWarping && tx == 0)
    {
        int4 endTriPixelIds = make_int4(ty * imgDim[0] + imgDim[0] - 1, ty * imgDim[0], (ty + 1) * imgDim[0] + imgDim[0] - 1, (ty + 1) * imgDim[0]);

        colB = inputColor[endTriPixelIds.y];
        colC = inputColor[endTriPixelIds.z];

        depB = inputDepth[endTriPixelIds.y];
        depC = inputDepth[endTriPixelIds.z];

        posB = inputPositions[endTriPixelIds.y];
        posC = inputPositions[endTriPixelIds.z];

        processTriangle(inputPositions[endTriPixelIds.x], posB, posC, inputColor[endTriPixelIds.x], colB, colC, inputDepth[endTriPixelIds.x], depB, depC, outputColor, outputDepthValidity);
        processTriangle(inputPositions[endTriPixelIds.w], posC, posB, inputColor[endTriPixelIds.w], colC, colB, inputDepth[endTriPixelIds.w], depC, depB, outputColor, outputDepthValidity);
    }
}

__global__ void initializeArrayWithEncodedValues(unsigned long long int* array, float depth, float validity, int size)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < size)
    {
        array[threadId] = encodeDepthAndValidity(depth, validity);
    }
}

__global__ void separateDepthAndValidityKernel(unsigned long long int* outputDepthValidity, float* outputDepth, float* validity, int size)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < size)
    {
        DepthAndValidity dv = decodeDepthAndValidity(outputDepthValidity[threadId]);

        outputDepth[threadId] = dv.depth;
        validity[threadId] = dv.validity;
    }
}


/**
 * @brief Transforms triangles from the input image to the output image based on depth and position information.
 *
 * @param inputColor Input color image as a 3-channel float matrix containing the YUV values.
 * @param inputDepth Input depth image as a single-channel float matrix containing depth values.
 * @param inputPositions Input positions image as a 2-channel float matrix containing the x and y coordinates.
 * @param outputSize The desired size of the output color and depth images.
 * @param baseDepth Base depth image as a single-channel float matrix.
 * @param baseValidity Base validity mask as a single-channel float matrix.
 * @param horizontalWrap A boolean flag indicating whether to process the image for horizontal wrapping.
 * @param devOutputColor Pointer to the output color buffer.
 * @param devOutputDepth Pointer to the output depth buffer.
 * @param devQuality Pointer to the quality buffer.
 * @param devValidity Pointer to the validity buffer.
 */
void transform_trianglesMethod
(
    float3* devInputColor, cv::Size inputSize, float* devInputDepth, float2* devInputPositions, cv::Size outputSize, bool horizontalWrap,
    float3*& devOutputColor, float*& devOutputDepth, float*& devValidity
)
{
    int imgWidth = inputSize.width;
    int imgHeight = inputSize.height;

    int outputImgWidth = outputSize.width;
    int outputImgHeight = outputSize.height;
    int outputImgSize = outputImgWidth * outputImgHeight;

    int hostImgDim[2] = { imgWidth, imgHeight };
    int hostOutputSize[2] = { outputImgWidth, outputImgHeight };

    unsigned long long int* devOutputDepthValidity;

    hipMalloc(&devOutputDepthValidity, outputImgSize * sizeof(unsigned long long int));
    hipMalloc(&devOutputColor, outputImgSize * sizeof(float3));
    hipMalloc(&devOutputDepth, outputImgSize * sizeof(float));
    hipMalloc(&devValidity, outputImgSize * sizeof(float));

    int blockSize = 256;
    int numBlocks = (outputImgSize + blockSize - 1) / blockSize;

    initializeArrayWithEncodedValues <<<numBlocks, blockSize>>> (devOutputDepthValidity, INFINITY, 0, outputImgSize);

    hipMemcpyToSymbol(HIP_SYMBOL(imgDim), &hostImgDim, 2 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(outputImgDim), &hostOutputSize, 2 * sizeof(int));

    int blockWidth = 16;
    int blockHeight = 8;
    dim3 gridDim((imgWidth - 1 + blockWidth) / blockWidth, (imgHeight - 1 + blockHeight) / blockHeight);
    dim3 blockDim(blockWidth, blockHeight);

    colorizeTriangleKernel <<<gridDim, blockDim>>> (devInputColor, devInputDepth, devInputPositions, devOutputColor, devOutputDepthValidity, horizontalWrap);

    hipDeviceSynchronize();

    separateDepthAndValidityKernel <<<numBlocks, blockSize>>> (devOutputDepthValidity, devOutputDepth, devValidity, outputImgSize);

    hipDeviceSynchronize();

    hipFree(devInputColor);
    hipFree(devInputDepth);
    hipFree(devInputPositions);
    hipFree(devOutputDepthValidity);

    hipError_t state = hipGetLastError();
    bool error = state != hipSuccess;
    if (error)
        throw std::runtime_error(hipGetErrorString(state));
}
